// 
// Copyright:
// 
// Copyright (c) 2024 Microsoft Research
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <cassert>
#include <cstddef>
#include <cstdint>
#include <chrono>
#include <fcntl.h>
#include <filesystem>
#include <omp.h>
#include <unistd.h>

#include "utils/gpu_data_types.h"
#include "utils/gpu_file_utils.h"
#include "utils/misc_utils.h"
#include "utils/gpu_comms.h"
#include "utils/gpu_mem.h"
#include "utils/hip/hip_runtime_api.h"
#include "utils/gpu_random.h"

#include "cnn.h"

#include <sytorch/backend/llama_base.h>
#include <sytorch/softmax.h>

u64 *gpuGenSoftmaxKey(int batchSz, int numClasses, u64 *d_mask_I, bool secfloat, LlamaBase<u64> *llama)
{
    Tensor4D<u64> inpMask(batchSz, numClasses, 1, 1);
    Tensor4D<u64> softmaxOpMask(batchSz, numClasses, 1, 1);
    size_t memSz = batchSz * numClasses * sizeof(u64);
    moveIntoCPUMem((u8 *)inpMask.data, (u8 *)d_mask_I, memSz, NULL);
    gpuFree(d_mask_I);
    if (secfloat)
    {
        softmax_secfloat(inpMask, softmaxOpMask, dcf::orca::global::scale, 1);
    }
    else
    {
        pirhana_softmax(inpMask, softmaxOpMask, dcf::orca::global::scale, 0);
    }
    d_mask_I = (u64 *)moveToGPU((u8 *)softmaxOpMask.data, memSz, NULL);
    return d_mask_I;
}

void genModelKey(dcf::orca::GPUModel<u64> *m, u8 **bufPtr, int party, AESGlobalContext *g, bool secfloat, LlamaBase<u64> *llama, int epoch)
{
    auto d_mask_I = randomGEOnGpu<u64>(m->inpSz, dcf::orca::global::bw);
    auto h_mask_I = (u64*) moveToCPU((u8*)d_mask_I, m->inpSz * sizeof(u64), NULL);
    printf("Generate Model Key\n");
    for (int i = 0; i < 10; i++){
        printf("h_mask_I[%d] = %lu\n", i, h_mask_I[i]);
    }
    u64 *d_mask_O = NULL;
    for (int i = 0; i < m->layers.size(); i++)
    {   
        d_mask_O = m->layers[i]->genForwardKey(bufPtr, party, d_mask_I, g);
        assert(d_mask_O != d_mask_I);
        gpuFree(d_mask_I);
        d_mask_I = d_mask_O;
    }
    d_mask_I = gpuGenSoftmaxKey(m->batchSz, m->classes, d_mask_I, secfloat, llama);
    for (int i = m->layers.size() - 1; i >= 0; i--)
    {
        d_mask_I = m->layers[i]->genBackwardKey(bufPtr, party, d_mask_I, g, epoch);
    }
}

void writeKeySz(std::string dir, std::string modelName, u64 keySz)
{
    makeDir(dir);
    std::ofstream keySzFile(dir + modelName + ".txt");
    keySzFile << keySz;
    keySzFile.close();
}

void dealerE2E(std::string modelName, int party, int epochs, int blocks, int blockSz, int batchSz, int H, int W, int C, bool secfloat, bool momentum, std::string keyDir, int sleepInt, std::string weightsMask = "", bool fake_offline = true)
{
    AESGlobalContext g;
    initAESContext(&g);
    initGPURandomness();
    initGPUMemPool();
    sytorch_init();
    // assert(epochs < 6);

    auto expName = modelName + "-" + std::to_string(epochs) + "e-" + std::to_string(blocks) + "b";
    auto trainingDir = "output/P" + std::to_string(party) + "/training/";
    auto lossDir = trainingDir + "loss/" + expName + "/";
    auto keySzDir = trainingDir + "keysize/";
    auto weightsDir = lossDir + "weights/";

    // assumes output/P0/training exists
    makeDir(trainingDir + "loss/");
    makeDir(lossDir);
    makeDir(weightsDir);
    makeDir(keySzDir);

    char one = 1;
    char two = 2;

    std::cout << "before getGPUModel" << std::endl;

    // load the model
    dcf::orca::GPUModel<u64> *m = getGPUModel<u64>(modelName, Tensor<u64>(nullptr, {(u64)batchSz, (u64)H, (u64)W, (u64)C}));
    std::cout << "after getGPUModel" << std::endl;
    m->setTrain(momentum);
    m->initWeights(weightsMask, false);

    char *zeros;
    size_t padding, bufSize = 8 * OneGB;
    u8 *startPtr, *curPtr, *tmpPtr1, *tmpPtr2;
    getAlignedBuf(&startPtr, bufSize);

    // initialize llama
    LlamaConfig::party = DEALER;
    auto llama = new LlamaBase<u64>();
    tmpPtr1 = (u8 *)malloc(OneGB);
    bool isServer = party + 2 == SERVER;
    llama->initDealer((char **)(isServer ? &curPtr : &tmpPtr2), (char **)(isServer ? &tmpPtr2 : &curPtr));
    
    std::string keyFile = keyDir + modelName + "_training_key" + std::to_string(party);
    int fd = openForWriting(keyFile + "_" + to_string(0) + "_" + to_string(0) + "_" + std::to_string(0) + ".dat");
    for (int l = 0; l < epochs; l++)
    {
        for (int k = 0; k < blocks; k++)
        {
            printf("Iteration=%u\n", l * blocks * blockSz + k * blockSz);
            for (int j = 0; j < blockSz; j++)
            {
                curPtr = startPtr;
                tmpPtr2 = tmpPtr1;
                genModelKey(m, &curPtr, party, &g, secfloat, (LlamaBase<u64> *)llama, l);
                if (l == 0 && k == 0 && j == 0)
                {
                    size_t keySz = curPtr - startPtr;
                    padding = 4096 - (keySz % 4096);
                    keySz += padding;
                    zeros = new char[padding];
                    memset(zeros, 0, padding);
                    writeKeySz(keySzDir, modelName, keySz);
                }
                memcpy(curPtr, zeros, padding);
                curPtr += padding;
                writeKeyBuf(fd, curPtr - startPtr, startPtr);
            }
            m->dumpWeights(weightsDir + "weights_mask_" + std::to_string(party) + "_" + to_string(l) + "_" + to_string(k) + "_" + std::to_string(blockSz-1) + ".dat");
        }
    }
    close(fd);
    delete[] zeros;
    destroyGPURandomness();
}


int main(int argc, char *argv[])
{
    int party = atoi(argv[1]);
    auto keyDir = std::string(argv[2]);

    omp_set_num_threads(32);
    int epochs = 1;
    int blocks =  46;
    int blockSz = 10; // 600;
    int batchSz = 128;
    dealerE2E("CNN2", party, epochs, blocks, blockSz, batchSz, 28, 28, 1, true, true, keyDir, 300, "", true);
    // dealerE2E("P-SecureML", party, epochs, blocks, blockSz, batchSz, 28, 28, 1, false, true, keyDir, 300, "", true);
    return 0;
}
